#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../runner/runner.h"

#define NUM_OF_GPU_THREADS 1024

int nearestPowerOf2(int N)
{
    int a = log2(N);
 
    if (pow(2, a) == N)
        return N;
 
    return pow(2, a + 1);
}

double cpu_time(void)
{
  double value;

  value = (double)clock() / (double)CLOCKS_PER_SEC;

  return value;
}

__global__ void prime_number(int* globalTotal, int n)
{
  int i;
  int j;
  __shared__ volatile int total[NUM_OF_GPU_THREADS];

  total[threadIdx.x] = 1;

  i = 2 + (threadIdx.x * gridDim.x + blockIdx.x);
  if(i <= n)
  {
    for (j = 2; j < i; j++)
    {
      if ((i % j) == 0)
      {
        total[threadIdx.x] = 0;
        break;
      }
    }
  }
  else
  {
    total[threadIdx.x] = 0;
  }

  __syncthreads();
  
  for (int iter = blockDim.x >> 1 ; iter > 32; iter >>= 1) 
  {
    if ( threadIdx.x < iter) 
    {
      total[threadIdx.x] += total[threadIdx.x + iter];
    } 
    __syncthreads();
  } 
  if (threadIdx.x < 32) total[threadIdx.x] += total[threadIdx.x + 32];
  if (threadIdx.x < 16) total[threadIdx.x] += total[threadIdx.x + 16];
  if (threadIdx.x < 8)  total[threadIdx.x] += total[threadIdx.x + 8];
  if (threadIdx.x < 4)  total[threadIdx.x] += total[threadIdx.x + 4];
  if (threadIdx.x < 2)  total[threadIdx.x] += total[threadIdx.x + 2];
  if (threadIdx.x < 1)  atomicAdd(globalTotal,total[threadIdx.x] + total[threadIdx.x + 1]);
}

void timestamp(void)
{
#define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  size_t len;
  time_t now;

  now = time(NULL);
  tm = localtime(&now);

  len = strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

  printf("%s\n", time_buffer);

  return;
#undef TIME_SIZE
}

void test(int n_lo, int n_hi, int n_factor);

int main(int argc, char *argv[])
{
  int n_factor;
  int n_hi;
  int n_lo;

  timestamp();
  printf("\n");
  printf("PRIME TEST\n");

  if (argc != 4)
  {
    n_lo = 1;
    n_hi = 131072;
    n_factor = 2;
  }
  else
  {
    n_lo = atoi(argv[1]);
    n_hi = atoi(argv[2]);
    n_factor = atoi(argv[3]);
  }

  test(n_lo, n_hi, n_factor);

  printf("\n");
  printf("PRIME_TEST\n");
  printf("  Normal end of execution.\n");
  printf("\n");
  timestamp();

  __runner__print();

  return 0;
}

void test(int n_lo, int n_hi, int n_factor)
{
  int i;
  int n;
  int primes;
  double ctime;

  printf("\n");
  printf("  Call PRIME_NUMBER to count the primes from 1 to N.\n");
  printf("\n");
  printf("         N        Pi          Time\n");
  printf("\n");

  n = n_lo;

  while (n <= n_hi)
  {
    int* globalTotal;
    primes = 0;

    int idealGrid = nearestPowerOf2(ceil(n/(float)NUM_OF_GPU_THREADS));
    if(idealGrid==1) idealGrid = 2;
    int idealBlock = nearestPowerOf2(n/(float)idealGrid);
    dim3 gridDim(idealGrid);
    dim3 blockDim(idealBlock);

    hipMalloc(&globalTotal, sizeof(int));
    hipMemcpy(globalTotal,&primes,sizeof(int),hipMemcpyHostToDevice);

    ctime = cpu_time();

    __runner__start();

    prime_number<<<gridDim,blockDim>>>(globalTotal,n);

    __runner__stop();

    ctime = cpu_time() - ctime;

    hipMemcpy(&primes,globalTotal,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(globalTotal);

    printf("  %8d  %8d  %14f\n", n, primes, ctime);
    n = n * n_factor;
  }

  return;
}
